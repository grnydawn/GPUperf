#include "hip/hip_runtime.h"
#include <stdio.h>
#include <nvToolsExt.h>

// Size of array
#define N (1 << 20)

// Kernel
__global__ void add_vectors(double *a, double *b, double *c)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < N)
		c[id] = a[id] + b[id];
}

// Main program
int main()
{
    nvtxRangePush("main");

	// Number of bytes to allocate for N doubles
	size_t bytes = N*sizeof(double);

	// Allocate memory for arrays A, B, and C on host
	double *A = (double*)malloc(bytes);
	double *B = (double*)malloc(bytes);
	double *C = (double*)malloc(bytes);

	// Allocate memory for arrays d_A, d_B, and d_C on device
    nvtxRangePush("hipMalloc");
	double *d_A, *d_B, *d_C;
	hipMalloc(&d_A, bytes);
	hipMalloc(&d_B, bytes);
	hipMalloc(&d_C, bytes);
    nvtxRangePop();

	// Fill host arrays A and B
	for(int i=0; i<N; i++)
	{
		A[i] = (double) rand();
		B[i] = (double) rand();
	}

	// Copy data from host arrays A and B to device arrays d_A and d_B
    nvtxRangePush("hipMemcpy H-to-D");
	hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice);
    nvtxRangePop();

	// Set execution configuration parameters
	//		thr_per_blk: number of CUDA threads per grid block
	//		blk_in_grid: number of blocks in grid
	int thr_per_blk = 256;
	int blk_in_grid = ceil( float(N) / thr_per_blk );

	// Launch kernel
    nvtxRangePush("kernel launch");
	add_vectors<<< blk_in_grid, thr_per_blk >>>(d_A, d_B, d_C);
    nvtxRangePop();

	// Copy data from device array d_C to host array C
    nvtxRangePush("hipMemcpy D-to-H");
	hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost);
    nvtxRangePop();

	// Verify results
	for(int i=0; i<N; i++)
	{
		if(C[i] != A[i]+B[i])
		{ 
			printf("\nError: value of C[%d] = %f instead of %f\n\n", i, C[i], A[i]+B[i]);
			exit(-1);
		}
	}	

	// Free CPU memory
	free(A);
	free(B);
	free(C);

	// Free GPU memory
    nvtxRangePush("hipFree");
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
    nvtxRangePop();

	printf("\n---------------------------\n");
	printf("__SUCCESS__\n");
	printf("---------------------------\n");
	printf("N                 = %d\n", N);
	printf("Threads Per Block = %d\n", thr_per_blk);
	printf("Blocks In Grid    = %d\n", blk_in_grid);
	printf("---------------------------\n\n");

    nvtxRangePop();

	return 0;
}
